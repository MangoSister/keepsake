#include "memory_low_level.cuh"
#include <cstdio>

#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#ifndef WIN32_LEAN_AND_MEAN
#define WIN32_LEAN_AND_MEAN
#endif
#ifndef NOMINMAX
#define NOMINMAX
#endif
#include <aclapi.h>
#include <sddl.h>
#include <windows.h>
#include <winternl.h>
#endif

namespace ksc
{

// https://github.com/NVIDIA/cuda-samples/tree/master/Samples/5_Domain_Specific/simpleVulkanMMAP

// Windows-specific LPSECURITYATTRIBUTES
void set_default_security_descriptor(hipMemAllocationProp *prop)
{
#if defined(__linux__)
    return;
#elif defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    static const char sddl[] = "D:P(OA;;GARCSDWDWOCCDCLCSWLODTWPRPCRFA;;;WD)";
    static OBJECT_ATTRIBUTES obj_attributes;
    static bool obj_attributes_configured = false;

    if (!obj_attributes_configured) {
        PSECURITY_DESCRIPTOR sec_desc;
        BOOL result = ConvertStringSecurityDescriptorToSecurityDescriptorA(sddl, SDDL_REVISION_1, &sec_desc, NULL);
        if (result == 0) {
            fprintf(stderr, "IPC failure: getDefaultSecurityDescriptor Failed! (%d)\n", GetLastError());
        }

        InitializeObjectAttributes(&obj_attributes, NULL, 0, NULL, sec_desc);

        obj_attributes_configured = true;
    }

    prop->win32HandleMetaData = &obj_attributes;
    return;
#endif
}

CudaShareableLowLevelMemory cuda_alloc_device_low_level(size_t size, int device)
{
    // `ipc_handle_type_flag` specifies the platform specific handle type this sample
    // uses for importing and exporting memory allocation. On Linux this sample
    // specifies the type as hipMemHandleTypePosixFileDescriptor meaning that
    // file descriptors will be used. On Windows this sample specifies the type as
    // hipMemHandleTypeWin32 meaning that NT HANDLEs will be used. The
    // ipc_handle_type_flag variable is a convenience variable and is passed by value
    // to individual requests.
#if defined(__linux__)
    hipMemAllocationHandleType ipc_handle_type_flag = hipMemHandleTypePosixFileDescriptor;
#else
    hipMemAllocationHandleType ipc_handle_type_flag = hipMemHandleTypeWin32;
#endif

    hipMemAllocationProp alloc_prop = {};
    alloc_prop.type = hipMemAllocationTypePinned;
    alloc_prop.location.type = hipMemLocationTypeDevice;
    alloc_prop.location.id = device;
    alloc_prop.win32HandleMetaData = nullptr;
    alloc_prop.requestedHandleTypes = hipMemHandleTypeWin32;

    // Windows-specific LPSECURITYATTRIBUTES is required when
    // hipMemHandleTypeWin32 is used. The security attribute defines the scope
    // of which exported allocations may be tranferred to other processes. For all
    // other handle types, pass NULL.
    set_default_security_descriptor(&alloc_prop);

    // Get the recommended granularity for m_cudaDevice.
    size_t granularity = 0;
    CU_CHECK(hipMemGetAllocationGranularity(&granularity, &alloc_prop, hipMemAllocationGranularityRecommended));

    size_t size_rounded = (((size + granularity - 1) / granularity) * granularity);

    // Reserve the required contiguous VA space for the allocations
    hipDeviceptr_t dptr = 0;
    CU_CHECK(hipMemAddressReserve(&dptr, size_rounded, granularity, 0U, 0));

    // Create the allocations as a pinned allocation on this device.
    // Create an allocation to store all the positions of points on the xy plane
    // and a second allocation which stores information if the corresponding
    // position is inside the unit circle or not.
    hipMemGenericAllocationHandle_t handle{};
    CU_CHECK(hipMemCreate(&handle, size_rounded, &alloc_prop, 0));

    // Export the allocation to a platform-specific handle. The type of handle
    // requested here must match the requestedHandleTypes field in the prop
    // structure passed to hipMemCreate. The handle obtained here will be passed to
    // vulkan to import the allocation.
    ShareableHandle shareable_handle{};
    CU_CHECK(hipMemExportToShareableHandle((void *)&shareable_handle, handle, ipc_handle_type_flag, 0));

    CU_CHECK(hipMemMap(dptr, size_rounded, 0, handle, 0));

    // Release the handles for the allocation. Since the allocation is currently
    // mapped to a VA range with a previous call to hipMemMap the actual freeing of
    // memory allocation will happen on an eventual call to hipMemUnmap. Thus the
    // allocation will be kept live until it is unmapped.
    CU_CHECK(hipMemRelease(handle));

    hipMemAccessDesc access_descriptor = {};
    access_descriptor.location.id = device;
    access_descriptor.location.type = hipMemLocationTypeDevice;
    access_descriptor.flags = hipMemAccessFlagsProtReadWrite;

    // Apply the access descriptor to the whole VA range. Essentially enables
    // Read-Write access to the range.
    CU_CHECK(hipMemSetAccess(dptr, size_rounded, &access_descriptor, 1));

    return CudaShareableLowLevelMemory{.dptr = dptr, .shareable_handle = shareable_handle, .size = size_rounded};
}

void cuda_free_device_low_level(const CudaShareableLowLevelMemory &m)
{
    if (!m.dptr) {
        return;
    }

    CU_CHECK(hipMemUnmap(m.dptr, m.size));

#if defined(__linux__)
    close(shHandle);
#else
    CloseHandle(m.shareable_handle);
#endif

    // Free the virtual address region.
    CU_CHECK(hipMemAddressFree(m.dptr, m.size));
}

} // namespace ksc