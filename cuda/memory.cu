#include "memory.cuh"

namespace ksc
{

void *cuda_alloc_managed(size_t size)
{
    void *ptr = nullptr;
    CUDA_CHECK(hipMallocManaged(&ptr, size));
    return ptr;
}

void cuda_free_managed(void *ptr) { CUDA_CHECK(hipFree(ptr)); }

} // namespace ksc