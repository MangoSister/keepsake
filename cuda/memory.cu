#include "memory.cuh"

namespace ksc
{

void *cuda_alloc_managed(size_t size)
{
    void *ptr = nullptr;
    CUDA_CHECK(hipMallocManaged(&ptr, size));
    return ptr;
}

void *cuda_alloc_device(size_t size)
{
    void *ptr = nullptr;
    CUDA_CHECK(hipMalloc(&ptr, size));
    return ptr;
}

void cuda_free(void *ptr) { CUDA_CHECK(hipFree(ptr)); }

} // namespace ksc