#include "memory.cuh"

namespace ksc
{

void *cuda_alloc_managed(size_t size)
{
    void *ptr = nullptr;
    cuda_check(hipMallocManaged(&ptr, size));
    return ptr;
}

void *cuda_alloc_device(size_t size)
{
    void *ptr = nullptr;
    cuda_check(hipMalloc(&ptr, size));
    return ptr;
}

void cuda_free(void *ptr) { cuda_check(hipFree(ptr)); }

} // namespace ksc